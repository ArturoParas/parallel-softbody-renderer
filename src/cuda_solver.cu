#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include "../include/cuda_solver.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#define THREADS_PER_BLOCK 4
#define NUM_BLOCKS 2
// #define MAX_CIRCLES THREADS_PER_BLOCK * NUM_BLOCKS

// #define MAX_RDONLY_PER_BLOCK 3
// #define MAX_RDONLY MAX_RDONLY_PER_BLOCK * NUM_BLOCKS

// #define MAX_NBORS_PER_CIRCLE 3
// #define MAX_NBORS MAX_NBORS_PER_CIRCLE * MAX_CIRCLES

// #define G -0.1
// #define DAMP 0.98
// #define DT 0.1
// #define DT2 DT * DT
// #define K 2.
// #define RL 1
// #define MASS 1

#define min(a,b) (a < b ? a : b)
#define max(a,b) (a > b ? a : b)
#define abs(a) (a < 0 ?  -a : a)

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess)
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

__constant__ GlobalConstants d_params;

__device__ __inline__ void move(const int t_idx, const int b_idx, const int d_off, const int d_idx)
{
  float3 prev_particle = d_params.prev_particles[d_idx];
  float3 curr_particle = d_params.curr_particles[d_idx];
  float3 next_particle = curr_particle;

  // Move according to current velocity
  next_particle.x += (curr_particle.x - prev_particle.x) * d_params.spring_damp;
  next_particle.y += (curr_particle.y - prev_particle.y) * d_params.spring_damp;
  next_particle.z += (curr_particle.z - prev_particle.z) * d_params.spring_damp;

  // Move according to acceleration
  for (int nbor = 0; nbor < d_params.max_nbors_per_particle; nbor++) {
    int n_off = d_params.nbor_map[(d_off + nbor) * d_params.max_nbors_per_particle + t_idx];
    if (n_off >= 0) {
      float3 nbor_particle = d_params.curr_particles[d_params.rdonly_nbors[d_off + n_off]];

      float dx = nbor_particle.x - curr_particle.x;
      float dy = nbor_particle.y - curr_particle.y;
      float dz = nbor_particle.z - curr_particle.z;

      float dist = sqrtf(dx * dx + dy * dy + dz * dz);
      // DEBUG
      if (dist < 1e-16) {
        printf("TOO CLOSE\n");
      }

      float norm_acc = d_params.spring_k * (dist - d_params.spring_rest_len) / dist
        / d_params.particle_mass;

      next_particle.x += norm_acc * dx * d_params.dt2_intermediate;
      next_particle.y += norm_acc * dy * d_params.dt2_intermediate;
      next_particle.z += (norm_acc + d_params.g) * dz * d_params.dt2_intermediate;
    }
  }
  __syncthreads();

  d_params.prev_particles[d_idx] = curr_particle;
  d_params.curr_particles[d_idx] = next_particle;
}

__global__ void update_kernel()
{
  int t_idx = threadIdx.x;
  int b_idx = blockIdx.x;
  int d_off = b_idx * blockDim.x;
  int d_idx = d_off + t_idx;

  move(t_idx, b_idx, d_off, d_idx);
}

void solver_update(GlobalConstants& h_params, float* h_curr_particles)
{
  update_kernel<<<h_params.num_blocks, h_params.particles_per_block>>>();
  cudaCheckError(hipDeviceSynchronize());
  cudaCheckError(hipMemcpy(h_curr_particles, h_params.curr_particles, h_params.max_particles * sizeof(float3), hipMemcpyDeviceToHost));

  printf("\n");
  for (int i = 0; i < h_params.max_particles * 3; i += 3) {
    printf("h_curr_particles[%d] = (%f, %f, %f)\n", i / 3, h_curr_particles[i], h_curr_particles[i + 1], h_curr_particles[i + 2]);
  }
}

void solver_setup(
  GlobalConstants& h_params, const float* h_curr_particles, const int16_t* h_rdonly_nbors,
  const int16_t* h_nbor_map)
{
  cudaCheckError(hipMalloc(&h_params.curr_particles, h_params.max_particles * sizeof(float3)));
  cudaCheckError(hipMalloc(&h_params.prev_particles, h_params.max_particles * sizeof(float3)));
  cudaCheckError(hipMalloc(&h_params.rdonly_nbors, h_params.max_rdonly * sizeof(int16_t)));
  cudaCheckError(hipMalloc(&h_params.nbor_map, h_params.max_nbors * sizeof(int16_t)));
  
  cudaCheckError(hipMemcpy(h_params.curr_particles, h_curr_particles, h_params.max_particles * sizeof(float3), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(h_params.prev_particles, h_curr_particles, h_params.max_particles * sizeof(float3), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(h_params.rdonly_nbors, h_rdonly_nbors, h_params.max_rdonly * sizeof(int16_t), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(h_params.nbor_map, h_nbor_map, h_params.max_nbors * sizeof(int16_t), hipMemcpyHostToDevice));

  cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_params), &h_params, sizeof(GlobalConstants)));

  printf("\n");
  for (int i = 0; i < 3 * h_params.max_particles; i += 3) {
    printf("h_curr_particles[%d] = (%f, %f, %f)\n", i / 3, h_curr_particles[i], h_curr_particles[i + 1], h_curr_particles[i + 2]);
  }
}

void solver_free(GlobalConstants& h_params)
{
  cudaCheckError(hipFree(h_params.curr_particles));
  cudaCheckError(hipFree(h_params.prev_particles));
  cudaCheckError(hipFree(h_params.rdonly_nbors));
  cudaCheckError(hipFree(h_params.nbor_map));
}
